
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <stdio.h>

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Incorrect input style, please do ./homework4 N" << std::endl;
        return 2;
    }

    int N = atoi(argv[1]);
    long int* arr = new long int[N];

    for (int i = 0; i < N; i++) {
        arr[i] = rand() % 1000 + 1;
    }

    long int* A_cpu = new long int[N];
    A_cpu[0] = 0;
    for (int i = 1; i < N; i++) {
        for (int j = 0; j < i; j++) {
            A_cpu[i] += arr[j];
        }
    }
    return 0;
}

