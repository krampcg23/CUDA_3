
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <ctime>
#include <assert.h>

__global__ void allPrefixSums (long int* A_gpu, long int* arr, int N) {
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    if (id == 0) return;
    if (id > N-1) return;
    for (int i = 0; i < id; i++) {
        A_gpu[id] += arr[i];
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Incorrect input style, please do ./homework4 N" << std::endl;
        return 2;
    }

    int N = atoi(argv[1]);
    long int* arr = new long int[N];

    for (int i = 0; i < N; i++) {
        arr[i] = rand() % 1000 + 1;
    }

    long int* A_cpu = new long int[N];

    // Sequential Code for all prefix sum
    A_cpu[0] = 0;
    for (int i = 1; i < N; i++) {
        A_cpu[i] += (arr[i-1] + A_cpu[i-1]);
    }

    long int* deviceA;
    hipMalloc(&deviceA, N * sizeof(long int));
    long int* deviceArr;
    hipMalloc(&deviceArr, N*sizeof(long int));
    hipMemcpy(deviceArr, arr, N*sizeof(long int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(1024, 1, 1);
    dim3 numBlocks(N / 1024 + 1, 1, 1);

    // Make the parallel call
    allPrefixSums<<<numBlocks, threadsPerBlock>>>(deviceA, deviceArr, N);

    long int* A_gpu = new long int[N];;
    hipMemcpy(A_gpu, deviceA, N*sizeof(long int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        assert(A_gpu[i] == A_cpu[i]);
    }
    printf("GPU Output Matches CPU Output\n");

    return 0;
}

